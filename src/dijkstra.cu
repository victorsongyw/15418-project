#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <limits.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

#define DIST_OFFSET 32
#define NODE_MASK 0xFFFFFFFF

#define THREADS_PER_BLOCK 512

extern uint N, M;
extern uint *nodes, *edges, *weights, *dists;

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n",
        hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
#else
#define cudaCheckError(ans) ans
#endif

// BASELINE VERSION ******************************** 
__global__ 
void baseline_Dijkstra_find_next_node(uint *nodes, uint *edges, uint *weights, uint *dists,
                              bool *finalized, unsigned long long int *min_dist_and_node, int num_nodes) 
{
    uint v = blockIdx.x * blockDim.x + threadIdx.x;
    if (v >= num_nodes) return;
    if (finalized[v]) return;
    unsigned long long int dist_and_node = ((unsigned long long int)dists[v] << DIST_OFFSET) 
                                            | (unsigned long long int)v;
    // dist is the upper bits, so we overwrite only if we have a smaller dist
    atomicMin(min_dist_and_node, dist_and_node); 
}

__global__ 
void baseline_Dijkstra_update_dists(uint *nodes, uint *edges, uint *weights, uint *dists,
                              bool *finalized, unsigned long long int *min_dist_and_node, int num_nodes) 
{
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint min_node = *min_dist_and_node & NODE_MASK;
    finalized[min_node] = true;

    // idx is the edge index for min_node's neighboring edges
    if (idx >= nodes[min_node+1] - nodes[min_node]) return;
    idx += nodes[min_node];

    uint v = edges[idx];
    if (!finalized[v] && dists[min_node] + weights[idx] < dists[v]) {
        dists[v] = dists[min_node] + weights[idx];
    }
}

// END BASELINE VERSION ******************************** 


// WARP-BASED VERSION ******************************** 

__global__ 
void warp_Dijkstra_find_next_node(uint *nodes, uint *edges, uint *weights, uint *dists,
                              bool *finalized, unsigned long long int *min_dist_and_node, int num_nodes) 
{
    uint v = blockIdx.x * blockDim.x + threadIdx.x;
    if (v >= num_nodes) return;

    // copy my work to shared memory
    __shared__ uint warp_dist[THREADS_PER_BLOCK];
    warp_dist[threadIdx.x] = dists[v];
    
    if (finalized[v]) return;
    
    unsigned long long int dist_and_node = ((unsigned long long int)warp_dist[threadIdx.x] << DIST_OFFSET) 
                                            | (unsigned long long int)v;
    // dist is the upper bits, so we overwrite only if we have a smaller dist
    atomicMin(min_dist_and_node, dist_and_node);
}

__global__ 
void warp_Dijkstra_update_dists(uint *nodes, uint *edges, uint *weights, uint *dists,
                              bool *finalized, unsigned long long int *min_dist_and_node, int num_nodes) 
{
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint min_node = *min_dist_and_node & NODE_MASK;
    finalized[min_node] = true;

    // idx is the edge index for min_node's neighboring edges
    if (idx >= nodes[min_node+1] - nodes[min_node]) return;
    idx += nodes[min_node];

    uint v = edges[idx];
    if (!finalized[v] && dists[min_node] + weights[idx] < dists[v])
        dists[v] = dists[min_node] + weights[idx];
}

// END WARP-BASED VERSION ******************************** 

// main function
void dijkstra_cuda(bool use_warp) 
{
    uint *device_nodes, *device_edges, *device_weights, *device_dists;
    bool *finalized;
    bool *device_finalized;
    unsigned long long int min_dist_and_node, *device_min_dist_and_node; // upper 32 bytes represent dist, lower 32 bytes represent node

    int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    cudaCheckError(hipMalloc(&device_nodes, (N+1) * sizeof(uint)));
    cudaCheckError(hipMalloc(&device_edges, M * sizeof(uint)));
    cudaCheckError(hipMalloc(&device_weights, M * sizeof(uint)));
    cudaCheckError(hipMalloc(&device_dists, N * sizeof(uint)));
    cudaCheckError(hipMalloc(&device_finalized, N * sizeof(bool)));
    cudaCheckError(hipMalloc(&device_min_dist_and_node, sizeof(unsigned long long int)));

    finalized = new bool[N];
    for (uint i = 0; i < N; i++)
        finalized[i] = false;

    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    cudaCheckError(hipMemcpy(device_nodes, nodes, (N+1) * sizeof(uint), hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(device_edges, edges, M * sizeof(uint), hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(device_weights, weights, M * sizeof(uint), hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(device_dists, dists, N * sizeof(uint), hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(device_finalized, finalized, N * sizeof(bool), hipMemcpyHostToDevice));

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess)
        fprintf(stderr, "WARNING: A CUDA error occured before launching: code=%d, %s\n", errCode, hipGetErrorString(errCode));

    // run kernel
    double kernelStartTime = CycleTimer::currentSeconds();

    for (uint i = 0; i < N-1; i++) 
    {
        min_dist_and_node = ULLONG_MAX;
        cudaCheckError(hipMemcpy(device_min_dist_and_node, &min_dist_and_node, sizeof(unsigned long long int), hipMemcpyHostToDevice));
        if (!use_warp) 
            baseline_Dijkstra_find_next_node<<<blocks, THREADS_PER_BLOCK>>>(
                device_nodes, device_edges, device_weights, device_dists, device_finalized, device_min_dist_and_node, N);
        else
            warp_Dijkstra_find_next_node<<<blocks, THREADS_PER_BLOCK>>>(
                device_nodes, device_edges, device_weights, device_dists, device_finalized, device_min_dist_and_node, N);
        
        cudaCheckError(hipDeviceSynchronize());

        if (!use_warp)
            baseline_Dijkstra_update_dists<<<blocks, THREADS_PER_BLOCK>>>(
                device_nodes, device_edges, device_weights, device_dists, device_finalized, device_min_dist_and_node, N);
        else
            warp_Dijkstra_update_dists<<<blocks, THREADS_PER_BLOCK>>>(
                device_nodes, device_edges, device_weights, device_dists, device_finalized, device_min_dist_and_node, N);
        
        cudaCheckError(hipDeviceSynchronize());
    }

    double kernelEndTime = CycleTimer::currentSeconds();

    hipMemcpy(dists, device_dists, N * sizeof(uint), hipMemcpyDeviceToHost);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    errCode = hipPeekAtLastError();
    if (errCode != hipSuccess)
        fprintf(stderr, "WARNING: A CUDA error occured after launching: code=%d, %s\n", errCode, hipGetErrorString(errCode));

    double overallDuration = endTime - startTime;
    double kernelDuration = kernelEndTime - kernelStartTime;
    if (!use_warp)
        printf("CUDA Baseline\n");
    else
        printf("CUDA Warp\n");
    
    printf("\tOverall: %.3f ms\n", 1000.f * overallDuration);
    printf("\tKernel: %.3f ms\n", 1000.f * kernelDuration);

    hipFree(device_nodes);
    hipFree(device_edges);
    hipFree(device_weights);
    hipFree(device_dists);
    hipFree(device_finalized);
    delete[] finalized;
}
