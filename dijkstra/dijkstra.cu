#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

extern float toBW(int bytes, float sec);
extern int N, M;
extern int *nodes, edges, weights, dists;

int* device_nodes, device_edges, device_weights, device_dists;

__global__ 
void baseline_Dijkstra_kernel() {
    
}

void baseline_Dijkstra() {

    int totalBytes = sizeof(int) * (N + M) * 2;

    // TODO: compute number of blocks and threads per block
    // const int threadsPerBlock = 512;
    // const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipMalloc(&device_nodes, (N+1) * sizeof(int));
    hipMalloc(&device_edges, M * sizeof(int));
    hipMalloc(&device_weights, M * sizeof(int));
    hipMalloc(&device_dists, N * sizeof(int));

    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    hipMemcpy(device_nodes, nodes, (N+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_edges, edges, M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_weights, weights, M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_dists, dists, N * sizeof(int), hipMemcpyHostToDevice);

    // run kernel
    double kernelStartTime = CycleTimer::currentSeconds();

    for (round = 0; round < N; round++) {
        baseline_Dijkstra_kernel<<<blocks, threadsPerBlock>>>(round);
        hipDeviceSynchronize();
    }

    double kernelEndTime = CycleTimer::currentSeconds();

    hipMemcpy(dists, device_dists, N * sizeof(int), hipMemcpyDeviceToHost);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    double kernelDuration = kernelEndTime - kernelStartTime;
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));
    printf("Kernel: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * kernelDuration, toBW(totalBytes, kernelDuration));

    hipFree(device_nodes);
    hipFree(device_edges);
    hipFree(device_weights);
    hipFree(device_dists);
}
